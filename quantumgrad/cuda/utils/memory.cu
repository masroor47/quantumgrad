#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void* allocate_gpu_memory(size_t size) {
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

extern "C" void free_gpu_memory(void* ptr) {
    hipFree(ptr);
}

extern "C" void copy_cpu_to_gpu(const void* host_data, void* device_data, size_t size) {
    printf("Device data: %p\n", device_data);
    // print size of data to be copied in bytes
    printf("Size: %ld\n", size);

    hipError_t err = hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    // print first 10 elements
    float* data = (float*)host_data;
    printf("CUDA _______________\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}

extern "C" void copy_gpu_to_cpu(const void* device_data, void* host_data, size_t size) {
    // print device data which is a pointer
    printf("Device data: %p\n", device_data);
    hipError_t err = hipMemcpy(host_data, device_data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    // print first 10 elements
    float* data = (float*)host_data;
    printf("CUDA _______________\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}