#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add_matrices(float *a, float *b, float *c, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void matmul_simple_(float *left, float *right, float *result, int lrows, int lcols, int rrows, int rcols) {
    // current row of left, current col of right
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < lrows && col < rcols) {
        float temp_sum = 0.0;
        // dot product
        for (int i = 0; i < lcols; i++) {
            temp_sum += left[row * lcols + i] * right[i * rcols + col];
        }
        result[row * rcols + col] = temp_sum;
    }
}

void cudaCheckError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("%s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

extern "C" void* allocate_gpu_memory(size_t size) {
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

extern "C" void free_gpu_memory(void* ptr) {
    hipFree(ptr);
}

extern "C" void copy_to_gpu(const void* host_data, void* device_data, size_t size) {
    hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);
}

extern "C" void copy_gpu_to_cpu(const void* device_data, void* host_data, size_t size) {
    hipMemcpy(host_data, device_data, size, hipMemcpyDeviceToHost);
}

extern "C" void add(float *a, float *b, float *c, int rows, int cols) {
    float *d_a, *d_b, *d_c;
    size_t size = rows * cols * sizeof(float);

    hipError_t err;

    printf("Allocating %.2f GB of GPU memory\n", (double)(size * 3) / (1024 * 1024 * 1024));
    
    err = hipMalloc((void**)&d_a, size);
    cudaCheckError(err, "CUDA malloc failed for d_a");
    
    err = hipMalloc((void**)&d_b, size);
    cudaCheckError(err, "CUDA malloc failed for d_b");

    err = hipMalloc((void**)&d_c, size);
    cudaCheckError(err, "CUDA malloc failed for d_c");

    err = hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    cudaCheckError(err, "CUDA memcpy failed for d_a");

    err = hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    cudaCheckError(err, "CUDA memcpy failed for d_b");

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    add_matrices<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, rows, cols);

    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(kernelErr));
    }

    hipMemcpy(c, d_c, rows*cols*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

extern "C" void matmul_simple(float *left, float *right, float *result, int lrows, int lcols, int rrows, int rcols) {
    if (lcols != rrows) {
        printf("Matrix dimensions mismatch for multiplication\n");
        return;
    }

    float *d_left, *d_right, *d_result;
    size_t left_size = lrows * lcols * sizeof(float);
    size_t right_size = rrows * rcols * sizeof(float);
    size_t result_size = lrows * rcols * sizeof(float);

    printf("Allocating %.2f GB of GPU memory\n", (double)(left_size + right_size + result_size) / (1024 * 1024 * 1024));

    hipError_t err;
    err = hipMalloc((void**)&d_left, left_size);
    cudaCheckError(err, "CUDA malloc failed for d_left");

    err = hipMalloc((void**)&d_right, right_size);
    cudaCheckError(err, "CUDA malloc failed for d_right");

    err = hipMalloc((void**)&d_result, result_size);
    cudaCheckError(err, "CUDA malloc failed for d_result");

    err = hipMemcpy(d_left, left, left_size, hipMemcpyHostToDevice);
    cudaCheckError(err, "CUDA memcpy failed for d_left");

    err = hipMemcpy(d_right, right, right_size, hipMemcpyHostToDevice);
    cudaCheckError(err, "CUDA memcpy failed for d_right");

    int BLOCK_SIZE = 16;
    int GRID_SIZE_ROWS = (int)ceil((float)lrows / BLOCK_SIZE);
    int GRID_SIZE_COLS = (int)ceil((float)rcols / BLOCK_SIZE);
    
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(GRID_SIZE_COLS, GRID_SIZE_ROWS);

    matmul_simple_<<<gridSize, blockSize>>>(d_left, d_right, d_result, lrows, lcols, rrows, rcols);

    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(kernelErr));
    }

    hipMemcpy(result, d_result, result_size, hipMemcpyDeviceToHost);

    hipFree(d_left);
    hipFree(d_right);
    hipFree(d_result);
}