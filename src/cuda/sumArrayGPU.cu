#include <stdio.h>
#include <hip/hip_runtime.h>



__global__ void my_cuda_kernel(int *input, int *output, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        output[tid] = input[tid] * 2;
    }
}

extern "C" void my_cuda_function(int *input, int *output, int size) {
    printf("in: %i\n", input[0]);
    printf("out: %i\n", output[0]);

    int *d_input, *d_output;
    hipMalloc((void**)&d_input, size * sizeof(int));
    hipMalloc((void**)&d_output, size * sizeof(int));

    hipMemcpy(d_input, input, size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    my_cuda_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, size);

    hipMemcpy(output, d_output, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}