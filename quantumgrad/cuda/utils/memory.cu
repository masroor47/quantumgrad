#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void* allocate_gpu_memory(size_t size) {
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

extern "C" void free_gpu_memory(void* ptr) {
    printf("Freeing GPU memory %p\n", ptr);
    hipFree(ptr);
}

extern "C" void copy_cpu_to_gpu(const void* host_data, void* device_data, size_t size) {
    printf("Device data: %p\n", device_data);
    printf("Host data: %p\n", host_data);
    // print size of data to be copied in bytes
    printf("Size: %ld\n", size);
    // print first 10 elements
    float* data = (float*)host_data;
    printf("CUDA _______________\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");

    hipError_t err = hipMemcpy(device_data, host_data, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
}

extern "C" void copy_gpu_to_cpu(const void* device_data, void* host_data, size_t size) {
    // print device data which is a pointer
    printf("Device data: %p\n", device_data);
    printf("Host data: %p\n", host_data);
    // print size of data to be copied in bytes
    printf("Size: %ld\n", size);
    hipError_t err = hipMemcpy(host_data, device_data, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    // print first 10 elements
    float* data = (float*)host_data;
    printf("CUDA _______________\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", data[i]);
    }
    printf("\n");
}