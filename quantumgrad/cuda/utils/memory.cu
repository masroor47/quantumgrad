#include <hip/hip_runtime.h>

extern "C" void* allocate_gpu_memory(size_t size) {
    void* ptr;
    hipMalloc(&ptr, size);
    return ptr;
}

extern "C" void free_gpu_memory(void* ptr) {
    hipFree(ptr);
}

extern "C" void copy_cpu_to_gpu(const void* host_data, void* device_data, size_t size) {
    hipMemcpy(device_data, host_data, size * sizeof(float), hipMemcpyHostToDevice);
}

extern "C" void copy_gpu_to_cpu(const void* device_data, void* host_data, size_t size) {
    hipMemcpy(host_data, device_data, size*sizeof(float), hipMemcpyDeviceToHost);
}