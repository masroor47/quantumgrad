
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_matrices(float *a, float *b, float *c, int rows, int cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        int idx = row * cols + col;
        c[idx] = a[idx] + b[idx];
    }
}

extern "C" void matrix_add(float *a, float *b, float *c, int rows, int cols) {
    float *d_a, *d_b, *d_c;

    printf("Allocating %.2f GB of GPU memory\n", (double)(rows * cols * sizeof(float) * 3) / (1024 * 1024 * 1024));
    hipMalloc((void**)&d_a, rows * cols * sizeof(float));
    hipMalloc((void**)&d_b, rows * cols * sizeof(float));
    hipMalloc((void**)&d_c, rows * cols * sizeof(float));

    hipMemcpy(d_a, a, rows*cols*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, rows*cols*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, rows*cols*sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (rows + threadsPerBlock.y - 1) / threadsPerBlock.y);

    add_matrices<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, rows, cols);

    hipError_t kernelErr = hipGetLastError();
    if (kernelErr != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(kernelErr));
    }

    hipMemcpy(c, d_c, rows*cols*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}